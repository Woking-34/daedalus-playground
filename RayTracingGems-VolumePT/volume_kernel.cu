#include "hip/hip_runtime.h"
//
// CUDA volume path tracing kernel implementation
//

#define _USE_MATH_DEFINES
#include <cmath>
#include "volume_kernel.h"

// 3d vector math utilities.
__device__ inline float3 operator+(const float3& a, const float3& b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}
__device__ inline float3 operator-(const float3& a, const float3& b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
__device__ inline float3 operator*(const float3& a, const float s)
{
    return make_float3(a.x * s, a.y * s, a.z * s);
}
__device__ inline float3 operator/(const float3& a, const float s)
{
    return make_float3(a.x / s, a.y / s, a.z / s);
}
__device__ inline void operator+=(float3& a, const float3& b)
{
    a.x += b.x; a.y += b.y; a.z += b.z;
}
__device__ inline void operator*=(float3& a, const float& s)
{
    a.x *= s; a.y *= s; a.z *= s;
}
__device__ inline float3 normalize(const float3 &d)
{
    const float inv_len = 1.0f / sqrtf(d.x * d.x + d.y * d.y + d.z * d.z);
    return make_float3(d.x * inv_len, d.y * inv_len, d.z * inv_len);
}
__device__ inline float dot(const float3 &u, const float3 &v)
{
    return u.x * v.x + u.y * v.y + u.z * v.z;
}

#include <hiprand/hiprand_kernel.h>
typedef hiprandStatePhilox4_32_10_t Rand_state;
#define rand(state) hiprand_uniform(state)

__device__ inline bool intersect_volume_box(
    float &tmin, const float3 &raypos, const float3 &raydir)
{
    const float x0 = (-0.5f - raypos.x) / raydir.x;
    const float y0 = (-0.5f - raypos.y) / raydir.y;
    const float z0 = (-0.5f - raypos.z) / raydir.z;
    const float x1 = ( 0.5f - raypos.x) / raydir.x;
    const float y1 = ( 0.5f - raypos.y) / raydir.y;
    const float z1 = ( 0.5f - raypos.z) / raydir.z;

    tmin = fmaxf(fmaxf(fmaxf(fminf(z0,z1), fminf(y0,y1)), fminf(x0,x1)), 0.0f);
    const float tmax = fminf(fminf(fmaxf(z0,z1), fmaxf(y0,y1)), fmaxf(x0,x1));
    return (tmin < tmax);
}

__device__ inline bool in_volume(
    const float3 &pos)
{
    return fmaxf(fabsf(pos.x), fmaxf(fabsf(pos.y), fabsf(pos.z))) < 0.5f;
}

__device__ inline float get_extinction(
    const Kernel_params &kernel_params,
    const float3 &p)
{
    if (kernel_params.volume_type == 0) {
        float3 pos = p + make_float3(0.5f, 0.5f, 0.5f);
        const unsigned int steps = 3;
        for (unsigned int i = 0; i < steps; ++i) {
            pos *= 3.0f;
            const int s =
                ((int)pos.x & 1) + ((int)pos.y & 1) + ((int)pos.z & 1);
            if (s >= 2)
                return 0.0f;
        }
        return kernel_params.max_extinction;
    } else {
        const float r = 0.5f * (0.5f - fabsf(p.y));
        const float a = (float)(M_PI * 8.0) * p.y;
        const float dx = (cosf(a) * r - p.x) * 2.0f;
        const float dy = (sinf(a) * r - p.z) * 2.0f;
        return powf(fmaxf((1.0f - dx * dx - dy * dy), 0.0f), 8.0f) * kernel_params.max_extinction;
    }
}

__device__ inline bool sample_interaction(
    Rand_state &rand_state,
    float3 &ray_pos,
    const float3 &ray_dir,
    const Kernel_params &kernel_params)
{
    float t = 0.0f;
    float3 pos;
    do {
        t -= logf(1.0f - rand(&rand_state)) / kernel_params.max_extinction;

        pos = ray_pos + ray_dir * t;
        if (!in_volume(pos))
            return false;
        
    } while (get_extinction(kernel_params, pos) < rand(&rand_state) * kernel_params.max_extinction);

    ray_pos = pos;
    return true;
}

__device__ inline float3 trace_volume(
    Rand_state &rand_state,
    float3 &ray_pos,
    float3 &ray_dir,
    const Kernel_params &kernel_params)
{
    float t0;
    float w = 1.0f;
    if (intersect_volume_box(t0, ray_pos, ray_dir)) {

        ray_pos += ray_dir * t0;

        unsigned int num_interactions = 0;
        while (sample_interaction(rand_state, ray_pos, ray_dir, kernel_params))
        {
            // Is the path length exeeded?
            if (num_interactions++ >= kernel_params.max_interactions)
                return make_float3(0.0f, 0.0f, 0.0f);

            w *= kernel_params.albedo;
            // Russian roulette absorption
            if (w < 0.2f) {
                if (rand(&rand_state) > w * 5.0f) {
                    return make_float3(0.0f, 0.0f, 0.0f);
                }
                w = 0.2f;
            }

            // Sample isotropic phase function.
            const float phi = (float)(2.0 * M_PI) * rand(&rand_state);
            const float cos_theta = 1.0f - 2.0f * rand(&rand_state);
            const float sin_theta = sqrtf(1.0f - cos_theta * cos_theta);
            ray_dir = make_float3(
                cosf(phi) * sin_theta,
                sinf(phi) * sin_theta,
                cos_theta);
        }
    }

    // Lookup environment.
    if (kernel_params.environment_type == 0) {
        const float f = (0.5f + 0.5f * ray_dir.y) * w;
        return make_float3(f, f, f);
    } else {
        const float4 texval = tex2D<float4>(
            kernel_params.env_tex,
            atan2f(ray_dir.z, ray_dir.x) * (float)(0.5 / M_PI) + 0.5f,
            acosf(fmaxf(fminf(ray_dir.y, 1.0f), -1.0f)) * (float)(1.0 / M_PI));
        return make_float3(texval.x * w, texval.y * w, texval.z * w);
    }
}

extern "C" __global__ void volume_rt_kernel(
    const Kernel_params kernel_params)
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= kernel_params.resolution.x || y >= kernel_params.resolution.y)
        return;

    // Initialize pseudorandom number generator (PRNG); assume we need no more than 4096 random numbers.
    const unsigned int idx = y * kernel_params.resolution.x + x;
    Rand_state rand_state;
    hiprand_init(idx, 0, kernel_params.iteration * 4096, &rand_state);

    // Trace from the pinhole camera.
    const float inv_res_x = 1.0f / (float)kernel_params.resolution.x;
    const float inv_res_y = 1.0f / (float)kernel_params.resolution.y;
    const float pr = (2.0f * ((float)x + rand(&rand_state)) * inv_res_x - 1.0f);
    const float pu = (2.0f * ((float)y + rand(&rand_state)) * inv_res_y - 1.0f);
    const float aspect = (float)kernel_params.resolution.y * inv_res_x;
    float3 ray_pos = kernel_params.cam_pos;
    float3 ray_dir = normalize(
        kernel_params.cam_dir * kernel_params.cam_focal + kernel_params.cam_right * pr + kernel_params.cam_up * aspect * pu);
    const float3 value = trace_volume(rand_state, ray_pos, ray_dir, kernel_params);

    // Accumulate.
    if (kernel_params.iteration == 0)
        kernel_params.accum_buffer[idx] = value;
    else
        kernel_params.accum_buffer[idx] = kernel_params.accum_buffer[idx] +
            (value - kernel_params.accum_buffer[idx]) / (float)(kernel_params.iteration + 1);
    
    // Update display buffer (simple Reinhard tonemapper + gamma).
    float3 val = kernel_params.accum_buffer[idx] * kernel_params.exposure_scale;
    val.x *= (1.0f + val.x * 0.1f) / (1.0f + val.x);
    val.y *= (1.0f + val.y * 0.1f) / (1.0f + val.y);
    val.z *= (1.0f + val.z * 0.1f) / (1.0f + val.z);
    const unsigned int r = (unsigned int)(255.0f *
                  fminf(powf(fmaxf(val.x, 0.0f), (float)(1.0 / 2.2)), 1.0f));
    const unsigned int g = (unsigned int)(255.0f *
                  fminf(powf(fmaxf(val.y, 0.0f), (float)(1.0 / 2.2)), 1.0f));
    const unsigned int b = (unsigned int)(255.0f *
                  fminf(powf(fmaxf(val.z, 0.0f), (float)(1.0 / 2.2)), 1.0f));
    kernel_params.display_buffer[idx] = 0xff000000 | (r << 16) | (g << 8) | b;
}

