#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#ifdef _WIN32
typedef unsigned int uint32_t;
typedef unsigned short uint16_t;
#endif

void SortCellIndices(uint32_t* cellIds, uint32_t* particleIndices, uint32_t numGrains)
{
	thrust::sort_by_key(thrust::device_ptr<uint32_t>(cellIds), 
		 				thrust::device_ptr<uint32_t>(cellIds+numGrains),
						thrust::device_ptr<uint32_t>(particleIndices));
}

void SortCellIndices(float* cellIds, uint32_t* particleIndices, uint32_t numGrains)
{
	thrust::sort_by_key(thrust::device_ptr<float>(cellIds), 
		 				thrust::device_ptr<float>(cellIds+numGrains),
						thrust::device_ptr<uint32_t>(particleIndices));
}
