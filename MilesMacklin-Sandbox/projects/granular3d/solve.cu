
#include <hip/hip_runtime.h>
#if 0

#include "solve.h"

#include <cuda.h>
#include <cuda_runtime_api.h>

#include <iostream>

#ifdef _WIN32
typedef unsigned int uint32_t;
//typedef unsigned short uint32_t;
#endif

using namespace std;

#define PROFILE 0
#define USE_GRID 1
#define USE_BOX_PRUNING 0

#define kRadius 0.1f
#define kMaxRadius (kRadius)
#define kInvCellEdge (0.5f/kMaxRadius)

#if USE_GRID
typedef uint32_t CellId;
#else
typedef float CellId;
#endif

struct GrainSystem
{
public:
	
	Vec3* mPositions;
	Vec3* mVelocities;
	float* mRadii;
	
	Vec3* mSortedPositions;
	Vec3* mSortedVelocities;
	float* mSortedRadii;

	Vec3* mNewVelocities;

	uint32_t* mCellStarts;
	uint32_t* mCellEnds;
	CellId* mCellIds;
	uint32_t* mIndices;

	uint32_t mNumGrains;
	GrainParams mParams;
};

#if PROFILE

struct CudaTimer
{
	CudaTimer(const char* name, cudaEvent_t start, cudaEvent_t stop, float& timer) : mTimer(timer), mName(name), mStart(start), mStop(stop)
	{
		cudaEventRecord(mStart, 0);
	}
	
	~CudaTimer()
	{
		cudaEventRecord(mStop, 0);
		cudaEventSynchronize(mStop);
		
		float elapsedTime;
		cudaEventElapsedTime(&elapsedTime, mStart, mStop);
		
		mTimer += elapsedTime;

		//cout << mName << " took: " << elapsedTime << endl;
	}
	
	float& mTimer;
	cudaEvent_t mStart;
	cudaEvent_t mStop;
	const char* mName;
};

#else
struct CudaTimer
{
	CudaTimer(const char*, cudaEvent_t, cudaEvent_t, float& ) {}
};
#endif

void SortCellIndices(uint32_t* cellIds, uint32_t* particleIndices, uint32_t numGrains);
void SortCellIndices(float* cellIds, uint32_t* particleIndices, uint32_t numGrains);

__device__ inline float sqr(float x) { return x*x; }


// calculate collision impulse
__device__ inline Vec3 CollisionImpulse(Vec3 va, Vec3 vb, float ma, float mb, Vec3 n, float d, float baumgarte, float friction, float overlap)
{
	// calculate relative velocity
	Vec3 vd = vb-va;
	
	// calculate relative normal velocity
	float vn = Dot(vd, n);
	
	Vec3 j = Vec3(0.0f, 0.0f, 0.0f);
	
	//if (vn < 0.0f)
	vn = min(vn, 0.0f);

	{
		// calculate relative tangential velocity
		Vec3 vt = vd - n*vn;	
		float vtsq = Dot(vt, vt);
		float rcpvt = rsqrtf(vtsq);// + 0.001f);
		
		// position bias
		float bias = baumgarte*min(d+overlap, 0.0f);

		Vec3 jn = -(vn + bias)*n;
		Vec3 jt = max(friction*vn*rcpvt, -1.0f)*vt;
		
		// crappy static friction
		if (fabsf(vtsq*rcpvt) < fabsf(friction*vn*2.0f) && vn < 0.0f)
			jt = -vt;				

		// total mass 
		float msum = ma + mb;
	
		// normal impulse
		j = (jn + jt)*mb/msum;
	}
	
	return j;
}

#if USE_GRID

const uint32_t kGridDim = 128;

// transform a world space coordinate into cell coordinate
__device__ inline uint32_t GridCoord(float x, float invCellEdge)
{
	// offset to handle negative numbers
	float l = x+1000.0f;
	
	uint32_t c = (uint32_t)(floorf(l*invCellEdge));
	return c;
}

__device__ inline uint32_t GridHash(int x, int y, int z)
{	
	uint32_t cx = x & (kGridDim-1);
	uint32_t cy = y & (kGridDim-1);
	uint32_t cz = z & (kGridDim-1);
	
	return cy*(kGridDim*kGridDim) + cx*kGridDim + cz;
}

/*
__device__ inline uint32_t GridHash(int x, int y, int z)
{
	const uint32_t p1 = 73856093; 
	const uint32_t p2 = 19349663;
	const uint32_t p3 = 53471161;
		
	uint32_t n = x*p1 ^ y*p2 ^ z*p3;
	return n&(kGridDim*kGridDim*kGridDim-1);
}
*/

__global__ void CreateCellIndices(const Vec3* positions, uint32_t* cellIds, uint32_t* particleIndices)
{
	uint32_t i = blockIdx.x*blockDim.x + threadIdx.x;

	Vec3 p = positions[i];
	
	cellIds[i] = GridHash(GridCoord(p.x, kInvCellEdge), GridCoord(p.y, kInvCellEdge), GridCoord(p.z, kInvCellEdge));
	particleIndices[i] = i;	
}

__global__ void CreateGrid(const uint32_t* cellIds, uint32_t* cellStarts, uint32_t* cellEnds, uint32_t numGrains)
{	
	uint32_t i = blockIdx.x*blockDim.x + threadIdx.x;
	
	// scan the particle-cell array to find the start and end
	uint32_t c = cellIds[i];
	
	if (i == 0)
	{
		cellStarts[c] = i;
	}
	else
	{
		uint32_t p = cellIds[i-1];

		if (c != p)
		{
			cellStarts[c] = i;
			cellEnds[p] = i;
		}
	}
	
	if (i == numGrains-1)
	{
		cellEnds[c] = i+1;
	}
}

__device__ inline Vec3 CollideSphere(Vec3 xa, Vec3 xb, Vec3 va, Vec3 vb, float ra, float rb, float baumgarte, float friction, float overlap)
{
	// distance to sphere
	Vec3 t = xa - xb;
	Vec3 j = Vec3(0.0f, 0.0f, 0.0f);

	float d = Dot(t, t);
	float rsum = ra + rb;
	float mtd = d - sqr(rsum);
			
	if (mtd < 0.0f)
	{
		Vec3 n = Vec3(0.0f, 1.0f, 0.0f);
				
		if (d > 0.0f)
		{
			float rcpDist = rsqrtf(d);

			n = t * rcpDist;
			d = d * rcpDist;
		}
				
		j = CollisionImpulse(vb, va, 1.0f, 1.0f, n, d-rsum, baumgarte, friction, overlap);
	}

	return j;
}

__device__ inline Vec3 CollideCell(int index, int cx, int cy, int cz, const uint32_t* cellStarts, const uint32_t* cellEnds, const uint32_t* indices,
				 const Vec3* positions, const Vec3* velocities, const float* radii, Vec3 x, Vec3 v, float r, float baumgarte, float friction, float overlap)
{
	Vec3 j = Vec3(0.0f, 0.0f, 0.0f);
	
	uint32_t cellIndex = GridHash(cx, cy, cz);
	uint32_t cellStart = cellStarts[cellIndex];
	uint32_t cellEnd = cellEnds[cellIndex];
	
	for (int i=cellStart; i < cellEnd; ++i)
	{
		uint32_t particleIndex = i;//indices[i];
		
		if (particleIndex != index)
		{		
			j += CollideSphere(x, positions[particleIndex], v, velocities[particleIndex], r, radii[particleIndex], baumgarte, friction, overlap);
		}		
	}
	
	return j;
}


#endif


__global__ void ReorderParticles(const Vec3* positions, const Vec3* velocities, const float* radii, Vec3* sortedPositions, Vec3* sortedVelocities, float* sortedRadii, const uint32_t* indices)
{
	uint32_t i = blockIdx.x*blockDim.x + threadIdx.x;
	
	int originalIndex = indices[i];

	sortedPositions[i] = positions[originalIndex];
	sortedVelocities[i] = velocities[originalIndex];
	sortedRadii[i] = radii[originalIndex];
}


__global__ void Collide(const Vec3* positions, const Vec3* velocities, const float* radii, const uint32_t* cellStarts, const uint32_t* cellEnds, const uint32_t* indices,
						Vec3* newVelocities, int numGrains, GrainParams params, float dt, float scale)
{
	const int index = blockIdx.x*blockDim.x + threadIdx.x;
		
	const Vec3 x = positions[index];
	const Vec3 v = velocities[index];
	const float  r = radii[index];

	Vec3 vd = Vec3(0.0f, 0.0f, 0.0f);

#if USE_GRID

	// collide particles
	int cx = GridCoord(x.x, kInvCellEdge);
	int cy = GridCoord(x.y, kInvCellEdge);
	int cz = GridCoord(x.z, kInvCellEdge);

	for (int k=cz-1; k <= cz+1; ++k)
	{
		for (int j=cy-1; j <= cy+1; ++j)
		{
			for (int i=cx-1; i <= cx+1; ++i)
			{
				vd += CollideCell(index, i, j, k, cellStarts, cellEnds, indices, positions, velocities, radii, x, v, r, params.mBaumgarte, params.mFriction, params.mOverlap);
			}
		}
	}
#endif

	// collide planes
	for (int i=0; i < params.mNumPlanes; ++i)
	{
		Vec4 p = params.mPlanes[i];
						
		// distance to plane
		float d = x.x*p.x + x.y*p.y + x.z*p.z + p.w;
			
		float mtd = d - r;
			
		if (mtd < 0.0f)
		{
			vd += CollisionImpulse(Vec3(0.0f, 0.0f, 0.0f), v, 0.0f, 1.0f, Vec3(p.x, p.y, p.z), mtd, params.mBaumgarte, 0.8f, params.mOverlap);
		}
	}
	
	// write back velocity
	newVelocities[index] = v + vd * scale;
}

__global__ void IntegrateForce(Vec3* velocities, Vec3 gravity, float damp, float dt)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;

	velocities[index] += (gravity - damp*velocities[index])*dt;
}


__global__ void IntegrateVelocity(Vec3* positions, Vec3* velocities, const Vec3* newVelocities, float dt)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;

	// x += v*dt
	velocities[index] = newVelocities[index];
	positions[index] += velocities[index]*dt;
}

/*
__global__ void PrintCellCounts(uint32_t* cellStarts, uint32_t* cellEnds)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;

	printf("%d\n", cellEnds[index]-cellStarts[index]);

}
*/

//------------------------------------------------------------------


GrainSystem* grainCreateSystem(int numGrains)
{
	GrainSystem* s = new GrainSystem();
	
	s->mNumGrains = numGrains;
	
	cudaMalloc(&s->mPositions, numGrains*sizeof(Vec3));
	cudaMalloc(&s->mVelocities, numGrains*sizeof(Vec3));
	cudaMalloc(&s->mNewVelocities, numGrains*sizeof(Vec3));
	cudaMalloc(&s->mRadii, numGrains*sizeof(float));
	
	cudaMalloc(&s->mSortedPositions, numGrains*sizeof(Vec3));
	cudaMalloc(&s->mSortedVelocities, numGrains*sizeof(Vec3));
	cudaMalloc(&s->mSortedRadii, numGrains*sizeof(float));

	// grid
#if USE_GRID
	cudaMalloc(&s->mCellStarts, kGridDim*kGridDim*kGridDim*sizeof(uint32_t));
	cudaMalloc(&s->mCellEnds, kGridDim*kGridDim*kGridDim*sizeof(uint32_t));
#endif

	cudaMalloc(&s->mCellIds, numGrains*sizeof(uint32_t));
	cudaMalloc(&s->mIndices, numGrains*sizeof(uint32_t));
	
	return s;
}

void grainDestroySystem(GrainSystem* s)
{
	cudaFree(s->mPositions);
	cudaFree(s->mVelocities);
	cudaFree(s->mNewVelocities);
	cudaFree(s->mRadii);	
	
	cudaFree(s->mSortedPositions);
	cudaFree(s->mSortedVelocities);
	cudaFree(s->mSortedRadii);	
	
#if USE_GRID
	cudaFree(s->mCellStarts);
	cudaFree(s->mCellEnds);
#endif
	cudaFree(s->mCellIds);
	cudaFree(s->mIndices);

	delete s;
}
void grainSetSprings(GrainSystem* s, const uint32_t* springIndices, const float* springLengths, uint32_t numSprings)
{
	/*
	s->mSpringIndices = (uint32_t*)malloc(numSprings*2*sizeof(uint32_t));
	s->mSpringLengths = (float*)malloc(numSprings*sizeof(float));

	memcpy(s->mSpringIndices, springIndices, numSprings*2*sizeof(uint32_t));
	memcpy(s->mSpringLengths, springLengths, numSprings*sizeof(float));
	
	s->mNumSprings = numSprings;
	*/
}


void grainSetPositions(GrainSystem* s, float* p, int n)
{
	cudaMemcpy(&s->mPositions[0], p, sizeof(Vec3)*n, cudaMemcpyHostToDevice);
}

void grainSetVelocities(GrainSystem* s, float* v, int n)
{
	cudaMemcpy(&s->mVelocities[0], v, sizeof(Vec3)*n, cudaMemcpyHostToDevice);	
}

void grainSetRadii(GrainSystem* s, float* r)
{
	cudaMemcpy(&s->mRadii[0], r, sizeof(float)*s->mNumGrains, cudaMemcpyHostToDevice);
}

void grainGetPositions(GrainSystem* s, float* p)
{
	cudaMemcpy(p, &s->mPositions[0], sizeof(Vec3)*s->mNumGrains, cudaMemcpyDeviceToHost);
}

void grainGetVelocities(GrainSystem* s, float* v)
{
	cudaMemcpy(v, &s->mVelocities[0], sizeof(Vec3)*s->mNumGrains, cudaMemcpyDeviceToHost);
}

void grainGetRadii(GrainSystem* s, float* r)
{
	cudaMemcpy(r, &s->mRadii[0], sizeof(float)*s->mNumGrains, cudaMemcpyDeviceToHost);
}

void grainSetParams(GrainSystem* s, GrainParams* params)
{
	//cudaMemcpy(s->mParams, params, sizeof(GrainParams), cudaMemcpyHostToDevice);
	s->mParams = *params;
}

void grainUpdateSystem(GrainSystem* s, float dt, int iterations, GrainTimers* timers)
{
	//iterations = 10;

	dt /= iterations;

	const int kNumThreadsPerBlock = 128;
	const int kNumBlocks = s->mNumGrains / kNumThreadsPerBlock;

	GrainParams params = s->mParams;
	params.mBaumgarte /= dt;
	
	cudaEvent_t start, stop;
	cudaEventCreate(&start);
	cudaEventCreate(&stop);

	cudaFuncSetCacheConfig(CreateCellIndices, cudaFuncCachePreferL1);
	cudaFuncSetCacheConfig(CreateGrid, cudaFuncCachePreferL1);
	cudaFuncSetCacheConfig(ReorderParticles, cudaFuncCachePreferL1);
	cudaFuncSetCacheConfig(IntegrateForce, cudaFuncCachePreferL1);
	cudaFuncSetCacheConfig(IntegrateVelocity, cudaFuncCachePreferL1);
	cudaFuncSetCacheConfig(Collide, cudaFuncCachePreferL1);

	for (int i=0; i < iterations; ++i)
	{
		{
			CudaTimer timer("CreateCellIndices", start, stop, timers->mCreateCellIndices);
			
			CreateCellIndices<<<kNumBlocks, kNumThreadsPerBlock>>>(s->mPositions, s->mCellIds, s->mIndices);
		}

		{ 
			CudaTimer timer("SortCellIndices", start, stop, timers->mSortCellIndices);
			
			SortCellIndices(s->mCellIds, s->mIndices, s->mNumGrains);
		}

#if USE_GRID
		{
			CudaTimer timer("CreateGrid", start, stop, timers->mCreateGrid);
			
			cudaMemset(s->mCellStarts, 0, sizeof(uint32_t)*kGridDim*kGridDim*kGridDim);
			cudaMemset(s->mCellEnds, 0, sizeof(uint32_t)*kGridDim*kGridDim*kGridDim);

			CreateGrid<<<kNumBlocks, kNumThreadsPerBlock>>>(s->mCellIds, s->mCellStarts, s->mCellEnds, s->mNumGrains);
		}
#endif

		{
			CudaTimer timer("ReorderParticles", start, stop, timers->mReorder);

			ReorderParticles<<<kNumBlocks, kNumThreadsPerBlock>>>(s->mPositions, s->mVelocities, s->mRadii, s->mSortedPositions, s->mSortedVelocities, s->mSortedRadii, s->mIndices);
		}
		
		//PrintCellCounts<<<kGridDim*kGridDim/kNumThreadsPerBlock, kNumThreadsPerBlock>>>(s->mCellStarts, s->mCellEnds);

		{
			float t;
			CudaTimer timer("Integrate Force", start, stop, t);

			IntegrateForce<<<kNumBlocks, kNumThreadsPerBlock>>>(s->mSortedVelocities, s->mParams.mGravity, s->mParams.mDamp, dt);
		}

		{
			CudaTimer timer("Collide", start, stop, timers->mCollide);
			
			float scale = 1;//float(i+1)/(iterations);

			Collide<<<kNumBlocks, kNumThreadsPerBlock>>>(s->mSortedPositions, s->mSortedVelocities, s->mSortedRadii, s->mCellStarts, s->mCellEnds, s->mIndices, s->mNewVelocities, s->mNumGrains, params, dt, scale);
		}

		{
			CudaTimer timer("Integrate", start, stop, timers->mIntegrate);
	
			IntegrateVelocity<<<kNumBlocks, kNumThreadsPerBlock>>>(s->mSortedPositions, s->mSortedVelocities, s->mNewVelocities, dt); 
		}
	
		swap(s->mSortedPositions, s->mPositions);
		swap(s->mSortedVelocities, s->mVelocities);
		swap(s->mSortedRadii, s->mRadii);
		
	}		


	cudaEventDestroy(start);
	cudaEventDestroy(stop);
}

#endif
