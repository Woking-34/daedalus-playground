#include "hip/hip_runtime.h"
#include <cutil_math.h>

__constant__ unsigned char c_perm_3d[256];
__shared__ unsigned char s_perm_3d[256]; // shared memory copy of permuation array
//unsigned char* d_perm_parts=NULL; // global memory copy of permutation array
// host version of permutation array
const static unsigned char h_perm[] = { 151, 160, 137, 91, 90, 15,
131, 13, 201, 95, 96, 53, 194, 233, 7, 225, 140, 36, 103, 30, 69, 142, 8, 99, 37, 240, 21, 10, 23,
190, 6, 148, 247, 120, 234, 75, 0, 26, 197, 62, 94, 252, 219, 203, 117, 35, 11, 32, 57, 177, 33,
88, 237, 149, 56, 87, 174, 20, 125, 136, 171, 168, 68, 175, 74, 165, 71, 134, 139, 48, 27, 166,
77, 146, 158, 231, 83, 111, 229, 122, 60, 211, 133, 230, 220, 105, 92, 41, 55, 46, 245, 40, 244,
102, 143, 54, 65, 25, 63, 161, 1, 216, 80, 73, 209, 76, 132, 187, 208, 89, 18, 169, 200, 196,
135, 130, 116, 188, 159, 86, 164, 100, 109, 198, 173, 186, 3, 64, 52, 217, 226, 250, 124, 123,
5, 202, 38, 147, 118, 126, 255, 82, 85, 212, 207, 206, 59, 227, 47, 16, 58, 17, 182, 189, 28, 42,
223, 183, 170, 213, 119, 248, 152, 2, 44, 154, 163, 70, 221, 153, 101, 155, 167, 43, 172, 9,
129, 22, 39, 253, 19, 98, 108, 110, 79, 113, 224, 232, 178, 185, 112, 104, 218, 246, 97, 228,
251, 34, 242, 193, 238, 210, 144, 12, 191, 179, 162, 241, 81, 51, 145, 235, 249, 14, 239, 107,
49, 192, 214, 31, 181, 199, 106, 157, 184, 84, 204, 176, 115, 121, 50, 45, 127, 4, 150, 254,
138, 236, 205, 93, 222, 114, 67, 29, 24, 72, 243, 141, 128, 195, 78, 66, 215, 61, 156, 180
};

__device__ inline int perm(int i) { return(c_perm_3d[i & 0xff]); }
__device__ inline float fade(float t) { return t * t * t * (t * (t * 6.f - 15.f) + 10.f); }
__device__ inline float lerpP(float t, float a, float b) { return a + t * (b - a); }
__device__ inline float grad(int hash, float x, float y, float z) {
	int h = hash & 15;                      // CONVERT LO 4 BITS OF HASH CODE
	float u = h<8 ? x : y,                 // INTO 12 GRADIENT DIRECTIONS.
		v = h<4 ? y : h == 12 || h == 14 ? x : z;
	return ((h & 1) == 0 ? u : -u) + ((h & 2) == 0 ? v : -v);
}

__device__ float inoise_parts2(float x, float y, float z) {
	int X = ((int)floorf(x)) & 255, // FIND UNIT CUBE THAT
		Y = ((int)floorf(y)) & 255,   // CONTAINS POINT.
		Z = ((int)floorf(z)) & 255;
	x -= floorf(x);               // FIND RELATIVE X,Y,Z
	y -= floorf(y);               // OF POINT IN CUBE.
	z -= floorf(z);
	float u = fade(x),            // COMPUTE FADE CURVES
		v = fade(y),                // FOR EACH OF X,Y,Z.
		w = fade(z);
	int A = perm(X) + Y, AA = perm(A) + Z, AB = perm(A + 1) + Z, // HASH COORDINATES OF
		B = perm(X + 1) + Y, BA = perm(B) + Z, BB = perm(B + 1) + Z; // THE 8 CUBE CORNERS,

	return lerpP(w, lerpP(v, lerpP(u, grad(perm(AA), x, y, z), // AND ADD
		grad(perm(BA), x - 1.f, y, z)),   // BLENDED
		lerpP(u, grad(perm(AB), x, y - 1.f, z),    // RESULTS
		grad(perm(BB), x - 1.f, y - 1.f, z))),     // FROM  8
		lerpP(v, lerpP(u, grad(perm(AA + 1), x, y, z - 1.f),  // CORNERS
		grad(perm(BA + 1), x - 1.f, y, z - 1.f)),    // OF CUBE
		lerpP(u, grad(perm(AB + 1), x, y - 1.f, z - 1.f),
		grad(perm(BB + 1), x - 1.f, y - 1.f, z - 1.f))));
#ifdef ORIG
	return(perm(X));
#endif

}

__device__ inline float noise1D(float x, float y, float z, int octaves,
	float lacunarity, float gain, float freq, float amp)
{
	float sum = 0.f;
	for (int i = 0; i<octaves; i++) {
		sum += inoise_parts2(x*freq, y*freq, z*freq)*amp;
		freq *= lacunarity;
		amp *= gain;
	}
	return sum;
}

__global__ void dampVelKernel(float3* vel, float damping, float dt, int nParts)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	int n = x;

	if (n<nParts) {

		vel[n] = (1 - damping*dt)*vel[n];

	}
}

__global__ void addGravityKernel(float3* vel, float3 gravityDir, float gravityStrength, float dt, int nParts)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	int n = x;

	if (n<nParts) {
		vel[n] = vel[n] + gravityDir*gravityStrength*dt;
	}
}

__global__ void addTurbulenceKernel(float3* vel, float3* pos,
								  float3 noiseAmp, float3 noiseOffset, int noiseOct, float noiseLac, float noiseFreq,
								  float dt, int nParts)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	int n = x;

	if (n<nParts) {

		float3 currVel = vel[n];
		
		float3 noise = make_float3(0,0,0);
		if (noiseAmp.x != 0)
			noise.x = noiseAmp.x*noise1D(pos[n].x+noiseOffset.x, pos[n].y+noiseOffset.y, pos[n].z+noiseOffset.z,
										noiseOct, noiseLac, 0.5, noiseFreq,1);
		if (noiseAmp.y != 0)
			noise.y  = noiseAmp.y*noise1D(pos[n].x+noiseOffset.x+2000, pos[n].y+noiseOffset.y, pos[n].z+noiseOffset.z,
										noiseOct, noiseLac, 0.5, noiseFreq,1);
		if (noiseAmp.x != 0)
			noise.z  += noiseAmp.z*noise1D(pos[n].x+noiseOffset.x+5000, pos[n].y+noiseOffset.y, pos[n].z+noiseOffset.z,
										noiseOct, noiseLac, 0.5, noiseFreq,1);

		vel[n] = vel[n] + noise*dt;

	}
}

__global__ void integrateParticlesKernel(float3* pos, float3* vel, float* age, float* life,
								  float4* colour, float opacity, float3 col1, float3 col2, float dt, int nParts)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	int n = x;

	if (n<nParts) {

		pos[n] = pos[n] + vel[n]*dt;

		age[n] = age[n] + dt;
		if (age[n] > life[n])
			age[n] = life[n];

		float ageNorm = age[n]/life[n];

		float3 col = lerp(col1,col2,ageNorm);
		float alpha = opacity*(1-pow(age[n]/life[n],2));

		colour[n] = make_float4(col.x,col.y,col.z,alpha);


	}
}

__global__ void initNewParticlesKernel(float3* pos, float3* vel, float* age, float* life,
										float3 initPos, float3 initVel, float radVelAmp,
										float3 noiseVelAmp, float3 noiseVelOffset, int noiseVelOct, float noiseVelLac, float noiseVelFreq,
										float initLife, float time, int nParts)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	int n = x;

	//__device__ inline float noise1D(float x, float y, float z, int octaves,
	//	     float lacunarity, float gain, float freq, float amp)

	if (n<nParts) {
						
		float3 radVel = radVelAmp * (pos[n] - initPos);	
		
		vel[n] = initVel + radVel;
							
		if (noiseVelAmp.x != 0)
			vel[n].x += noiseVelAmp.x*noise1D(pos[n].x+noiseVelOffset.x, pos[n].y+noiseVelOffset.y, pos[n].z+noiseVelOffset.z,
										noiseVelOct, noiseVelLac, 0.5, noiseVelFreq,1);
			
		if (noiseVelAmp.y != 0)
			vel[n].y  += noiseVelAmp.y*noise1D(pos[n].x+noiseVelOffset.x+2000, pos[n].y+noiseVelOffset.y, pos[n].z+noiseVelOffset.z,
										noiseVelOct, noiseVelLac, 0.5, noiseVelFreq,1);
	
		if (noiseVelAmp.x != 0)
			vel[n].z  += noiseVelAmp.z*noise1D(pos[n].x+noiseVelOffset.x+5000, pos[n].y+noiseVelOffset.y, pos[n].z+noiseVelOffset.z,
										noiseVelOct, noiseVelLac, 0.5, noiseVelFreq,1);
	
		age[n] = 0.0;
	}

	
}

__global__ void resetParticlesKernel(float3* pos, float3* vel, float* age, float* life, int nParts)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	int n = x;

	if (n<nParts) {
		pos[n] = make_float3(0.0,0.0,0.0);
		vel[n] =  make_float3(0.0,0.0,0.0);
		age[n] = 1.0;
		life[n] = 1.0;
	}
}


extern "C" void dampVelCu(float3* vel, float damping, float dt, int nParts){

	int nthreads = min(256, nParts);
	int nBlocks = nParts/nthreads + (!(nParts%nthreads)?0:1);
    dim3 blocks(nBlocks, 1,1); dim3 threads(nthreads, 1, 1);

	dampVelKernel<<< blocks, threads>>>(vel, damping, dt, nParts);
}

extern "C" void addGravityCu(float3* vel, float3 gravityDir, float gravityStrength, float dt, int nParts){

	int nthreads = min(256, nParts);
	int nBlocks = nParts/nthreads + (!(nParts%nthreads)?0:1);
    dim3 blocks(nBlocks, 1,1); dim3 threads(nthreads, 1, 1);

	addGravityKernel<<< blocks, threads>>>(vel, gravityDir, gravityStrength, dt, nParts);
}

extern "C" void addTurbulenceCu(float3* vel, float3* pos, float3 noiseAmp, float3 noiseOffset,
								int noiseOct, float noiseLac, float noiseFreq, float dt, int nParts){

	int nthreads = min(256, nParts);
	int nBlocks = nParts/nthreads + (!(nParts%nthreads)?0:1);
    dim3 blocks(nBlocks, 1,1); dim3 threads(nthreads, 1, 1);

	addTurbulenceKernel<<< blocks, threads>>>(vel, pos, noiseAmp, noiseOffset, noiseOct, noiseLac, noiseFreq, dt, nParts);
}

extern "C" void integrateParticlesCu(float3* pos, float3* vel, float* age, float* life,
								  float4* colour, float opacity, float3 col1, float3 col2,
									float dt, int nParts){

	int nthreads = min(256, nParts);
	int nBlocks = nParts/nthreads + (!(nParts%nthreads)?0:1);
    dim3 blocks(nBlocks, 1,1); dim3 threads(nthreads, 1, 1);

	integrateParticlesKernel<<< blocks, threads>>>(pos, vel, age, life, colour, opacity, col1, col2, dt, nParts);

}


extern "C" void resetParticlesCu(float3* pos, float3* vel, float* age, float* life, int nParts) {

	int nthreads = min(256, nParts);

	int nBlocks = nParts/nthreads + (!(nParts%nthreads)?0:1);

    dim3 blocks(nBlocks, 1,1);
    dim3 threads(nthreads, 1, 1);

    resetParticlesKernel<<< blocks, threads>>>(pos, vel, age, life, nParts);
}

extern "C" void initNewParticlesCu(float3* pos, float3* vel, float* age, float* life,
									float3 initPos, float3 initVel, float radVelAmp, 
									float3 noiseVelAmp, float3 noiseVelOffset, int noiseVelOct, float noiseVelLac, float noiseVelFreq,
									float initLife, float time, int nParts) {

	int nthreads = min(256, nParts);

	int nBlocks = nParts/nthreads + (!(nParts%nthreads)?0:1);

    dim3 blocks(nBlocks, 1,1);
    dim3 threads(nthreads, 1, 1);

	hipMemcpyToSymbol(HIP_SYMBOL(c_perm_3d), h_perm, sizeof(h_perm),0,hipMemcpyHostToDevice );

    initNewParticlesKernel<<< blocks, threads>>>(pos, vel, age, life, initPos, initVel, radVelAmp,
												noiseVelAmp, noiseVelOffset, noiseVelOct, noiseVelLac, noiseVelFreq,
												initLife, time, nParts);
}