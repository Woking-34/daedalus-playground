#include "hip/hip_runtime.h"
//perlinKernelPBO.cu (Rob Farber)
#include <cutil_math.h>
//#include <cutil_inline.h>

__constant__ unsigned char c_perm_3d[256];
__shared__ unsigned char s_perm_3d[256]; // shared memory copy of permuation array
//unsigned char* d_perm_parts=NULL; // global memory copy of permutation array
// host version of permutation array
const static unsigned char h_perm[] = {151,160,137,91,90,15,
   131,13,201,95,96,53,194,233,7,225,140,36,103,30,69,142,8,99,37,240,21,10,23,
   190, 6,148,247,120,234,75,0,26,197,62,94,252,219,203,117,35,11,32,57,177,33,
   88,237,149,56,87,174,20,125,136,171,168, 68,175,74,165,71,134,139,48,27,166,
   77,146,158,231,83,111,229,122,60,211,133,230,220,105,92,41,55,46,245,40,244,
   102,143,54, 65,25,63,161, 1,216,80,73,209,76,132,187,208, 89,18,169,200,196,
   135,130,116,188,159,86,164,100,109,198,173,186, 3,64,52,217,226,250,124,123,
   5,202,38,147,118,126,255,82,85,212,207,206,59,227,47,16,58,17,182,189,28,42,
   223,183,170,213,119,248,152,2,44,154,163, 70,221,153,101,155,167, 43,172,9,
   129,22,39,253, 19,98,108,110,79,113,224,232,178,185, 112,104,218,246,97,228,
   251,34,242,193,238,210,144,12,191,179,162,241, 81,51,145,235,249,14,239,107,
   49,192,214, 31,181,199,106,157,184,84,204,176,115,121,50,45,127, 4,150,254,
   138,236,205,93,222,114,67,29,24,72,243,141,128,195,78,66,215,61,156,180
   };

__device__ inline int perm(int i) { return(c_perm_3d[i&0xff]); }
__device__ inline float fade(float t) { return t * t * t * (t * (t * 6.f - 15.f) + 10.f); }
__device__ inline float lerpP(float t, float a, float b) { return a + t * (b - a); }
__device__ inline float grad(int hash, float x, float y, float z) {
  int h = hash & 15;                      // CONVERT LO 4 BITS OF HASH CODE
  float u = h<8 ? x : y,                 // INTO 12 GRADIENT DIRECTIONS.
    v = h<4 ? y : h==12||h==14 ? x : z;
  return ((h&1) == 0 ? u : -u) + ((h&2) == 0 ? v : -v);
}

__device__ float inoise_parts2__(float x, float y, float z) {
  int X = ((int)floorf(x)) & 255, // FIND UNIT CUBE THAT
    Y = ((int)floorf(y)) & 255,   // CONTAINS POINT.
    Z = ((int)floorf(z)) & 255;
  x -= floorf(x);               // FIND RELATIVE X,Y,Z
  y -= floorf(y);               // OF POINT IN CUBE.
  z -= floorf(z);
  float u = fade(x),            // COMPUTE FADE CURVES
    v = fade(y),                // FOR EACH OF X,Y,Z.
    w = fade(z);
  int A = perm(X)+Y, AA = perm(A)+Z, AB = perm(A+1)+Z, // HASH COORDINATES OF
    B = perm(X+1)+Y, BA = perm(B)+Z, BB = perm(B+1)+Z; // THE 8 CUBE CORNERS,
  
  return lerpP(w, lerpP(v, lerpP(u, grad(perm(AA), x  , y  , z   ), // AND ADD
				 grad(perm(BA), x-1.f, y  , z   )),   // BLENDED
			lerpP(u, grad(perm(AB), x  , y-1.f, z   ),    // RESULTS
			      grad(perm(BB), x-1.f, y-1.f, z   ))),     // FROM  8
	       lerpP(v, lerpP(u, grad(perm(AA+1), x  , y  , z-1.f ),  // CORNERS
			      grad(perm(BA+1), x-1.f, y  , z-1.f )),    // OF CUBE
		     lerpP(u, grad(perm(AB+1), x  , y-1.f, z-1.f ),
			   grad(perm(BB+1), x-1.f, y-1.f, z-1.f ))));
#ifdef ORIG
  return(perm(X));
#endif
 
}

__device__ inline float noise1D(float x, float y, float z, int octaves,
		     float lacunarity, float gain, float freq, float amp)
{
  float sum = 0.f;  
  for(int i=0; i<octaves; i++) {
	  sum += inoise_parts2__(x*freq, y*freq, z*freq)*amp;
    freq *= lacunarity;
    amp *= gain;
  }
  return sum;
}